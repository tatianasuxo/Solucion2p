#include "hip/hip_runtime.h"
#include "TDES.cuh"
#include "Chunk.cuh"
#include <Timer.hpp>

#include <hip/hip_runtime.h>

#include <iostream>

namespace CUDA {

__global__ void kernelEncode(TDESCA::chunk64* keys, TDESCA::chunk64* dataIn,
                             unsigned int threadCount, TDESCA::chunk64* dataOut)
{
    int ind = threadIdx.x + blockIdx.x * threadCount;
    TDESCA::TDES cipher;
    dataOut[ind] = cipher.Encode(keys[0], keys[1], keys[2], dataIn[ind]);
}

__global__ void kernelDecode(TDESCA::chunk64* keys, TDESCA::chunk64* dataIn,
                             unsigned int threadCount, TDESCA::chunk64* dataOut)
{
    int ind = threadIdx.x + blockIdx.x * threadCount;
    TDESCA::TDES cipher;
    dataOut[ind] = cipher.Decode(keys[0], keys[1], keys[2], dataIn[ind]);
}

void CudaEncode(TDESCA::chunk64 key1, TDESCA::chunk64 key2,
                TDESCA::chunk64 key3, TDESCA::chunk64* dataIn,
                unsigned int chunkCount, unsigned int threadsNum,
                TDESCA::chunk64* dataOut, double* timeOut)
{
    Timer timer;

    TDESCA::chunk64* cudaDataIn;
    TDESCA::chunk64* cudaDataOut;
    TDESCA::chunk64* cudaKeys;

    hipMalloc(&cudaDataIn, chunkCount * sizeof(TDESCA::chunk64));
    hipMalloc(&cudaKeys, 3 * sizeof(TDESCA::chunk64));
    hipMalloc(&cudaDataOut, chunkCount * sizeof(TDESCA::chunk64));

    TDESCA::chunk64 keys[] = { key1, key2, key3 };

    hipMemcpy(cudaDataIn, dataIn, chunkCount * sizeof(TDESCA::chunk64), hipMemcpyHostToDevice);
    hipMemcpy(cudaKeys, keys, 3 * sizeof(TDESCA::chunk64), hipMemcpyHostToDevice);

    unsigned int blockCount = chunkCount / threadsNum;

    timer.start();
    kernelEncode<<<blockCount, threadsNum>>>(cudaKeys, cudaDataIn, threadsNum, cudaDataOut);
    hipDeviceSynchronize();
    *timeOut = timer.stopNs();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cout << "CUDA Kernel failed to launch with error code " << err << std::endl;
        *timeOut = -1.0;
    }
    else
        err = hipMemcpy(dataOut, cudaDataOut, chunkCount * sizeof(TDESCA::chunk64), hipMemcpyDeviceToHost);

    hipFree(cudaDataIn);
    hipFree(cudaKeys);
    hipFree(cudaDataOut);
}

void CudaDecode(TDESCA::chunk64 key1, TDESCA::chunk64 key2,
                TDESCA::chunk64 key3, TDESCA::chunk64* dataIn,
                unsigned int chunkCount, unsigned int threadsNum,
                TDESCA::chunk64* dataOut, double* timeOut)
{
    Timer timer;

    TDESCA::chunk64* cudaDataIn;
    TDESCA::chunk64* cudaDataOut;
    TDESCA::chunk64* cudaKeys;
    hipMalloc(&cudaDataIn, chunkCount * sizeof(TDESCA::chunk64));
    hipMemcpy(cudaDataIn, dataIn, chunkCount * sizeof(TDESCA::chunk64), hipMemcpyHostToDevice);

    hipMalloc(&cudaKeys, 3 * sizeof(TDESCA::chunk64));
    TDESCA::chunk64 keys[] = { key1, key2, key3 };
    hipMemcpy(cudaKeys, keys, 3 * sizeof(TDESCA::chunk64), hipMemcpyHostToDevice);

    hipMalloc(&cudaDataOut, chunkCount * sizeof(TDESCA::chunk64));

    unsigned int blockCount = chunkCount / threadsNum;

    timer.start();
    kernelDecode<<<blockCount, threadsNum>>>(cudaKeys, cudaDataIn, threadsNum, cudaDataOut);
    hipDeviceSynchronize();
    *timeOut = timer.stopNs();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cout << "CUDA Kernel failed to launch with error code " << err << std::endl;
        *timeOut = -1.0;
    }
    else
        hipMemcpy(dataOut, cudaDataOut, chunkCount * sizeof(TDESCA::chunk64), hipMemcpyDeviceToHost);

    hipFree(cudaDataIn);
    hipFree(cudaDataOut);
    hipFree(cudaKeys);
}

} // namespace CUDA